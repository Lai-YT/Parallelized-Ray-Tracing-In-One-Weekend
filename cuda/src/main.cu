#include "hip/hip_runtime.h"
//==============================================================================================
// Originally written in 2016 by Peter Shirley <ptrshrl@gmail.com>
//
// To the extent possible under law, the author(s) have dedicated all copyright and related and
// neighboring rights to this software to the public domain worldwide. This software is
// distributed without any warranty.
//
// You should have received a copy (see file COPYING.txt) of the CC0 Public Domain Dedication
// along with this software. If not, see <http://creativecommons.org/publicdomain/zero/1.0/>.
//==============================================================================================

#include "rtweekend.h"

#include "camera.h"
#include "hittable_list.h"
#include "material.h"
#include "sphere.h"
#include "hip/hip_runtime_api.h"
#include "def.h"

#include <cstdlib>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

__device__ color ray_color(const ray& r, const hittable_list& world, int depth, hiprandState* rand_state) {
    auto curr_ray = r;
    auto curr_attenuation = color(1, 1, 1);
    for (int i = 0; i < depth; i++) {
        hit_record rec;
        if (world.hit(curr_ray, 0.001, infinity, rec)) {
            ray scattered;
            color attenuation;
            if (rec.mat_ptr->scatter(curr_ray, rec, attenuation, scattered, rand_state)) {
                curr_attenuation = curr_attenuation * attenuation;
                curr_ray = scattered;
            } else {
                return color(0, 0, 0);
            }
        } else {
            vec3 unit_direction = unit_vector(r.direction());
            auto t = 0.5*(unit_direction.y() + 1.0);
            auto c = (1.0-t)*color(1.0, 1.0, 1.0) + t*color(0.5, 0.7, 1.0);
            return c * curr_attenuation;
        }
    }
    // If we've exceeded the ray bounce limit, no more light is gathered.
    return color(0, 0, 0);
}

// The scene is set up by on the GPU.
__global__ void random_scene(hittable_list* world, int* rand_nums, std::size_t num_rand_nums) {
    // The initialization is performed only on the first thread.
    // No effect if the kernel function is called with both thread size and block size being 1.
    if (threadIdx.x + blockIdx.x != 0) {
        return;
    }

    auto map_width = MAP_SIZE;
    auto half_map_width = map_width / 2;

    // While initializing the world, we use the random numbers passed from the CPU.
    int rand_num_idx = 0;

    auto ground_material = new lambertian(color(0.5, 0.5, 0.5));
    world->add(new sphere(point3(0,-1000,0), 1000, ground_material));

    for (int a = -half_map_width; a < half_map_width; a++) {
        for (int b = -half_map_width; b < half_map_width; b++) {
            auto choose_mat = random_double_s(rand_nums[rand_num_idx++]);
            // XXX: Since we're using GCC as our host compiler, its order of
            // evaluation appears to be right-to-left. The order of evaluation is
            // unspecified in C++. Changing the host compiler to Clang or MSVC will
            // likely break this, as well as on other platforms.
            auto e2 = b + 0.9*random_double_s(rand_nums[rand_num_idx++]);
            auto e1 = 0.2;
            auto e0 = a + 0.9*random_double_s(rand_nums[rand_num_idx++]);
            point3 center(e0, e1, e2);

            if ((center - point3(4, 0.2, 0)).length() > 0.9) {
                material* sphere_material;

                if (choose_mat < 0.8) {
                    // diffuse
                    auto albedo = color::random_s(rand_nums, rand_num_idx) * color::random_s(rand_nums, rand_num_idx);
                    sphere_material = new lambertian(albedo);
                    world->add(new sphere(center, 0.2, sphere_material));
                } else if (choose_mat < 0.95) {
                    // metal
                    auto albedo = color::random_s(0.5, 1, rand_nums, rand_num_idx);
                    auto fuzz = random_double_s(0, 0.5, rand_nums[rand_num_idx++]);
                    sphere_material = new metal(albedo, fuzz);
                    world->add(new sphere(center, 0.2, sphere_material));
                } else {
                    // glass
                    sphere_material = new dielectric(1.5);
                    world->add(new sphere(center, 0.2, sphere_material));
                }
            }
        }
    }

    auto material1 = new dielectric(1.5);
    world->add(new sphere(point3(0, 1, 0), 1.0, material1));

    auto material2 = new lambertian(color(0.4, 0.2, 0.1));
    world->add(new sphere(point3(-4, 1, 0), 1.0, material2));

    auto material3 = new metal(color(0.7, 0.6, 0.5), 0.0);
    world->add(new sphere(point3(4, 1, 0), 1.0, material3));
}

__global__ void render(vec3* buffer, int image_width, int image_height,
                       const hittable_list* world, camera cam, int max_depth,
                       int samples_per_pixel, hiprandState* rand_states) {
    const auto j = threadIdx.y + blockDim.y * blockIdx.y;
    const auto i = threadIdx.x + blockDim.x * blockIdx.x;
    // We may be launching more threads than necessary.
    // Ignore those threads.
    if ((i >= image_width) || (j >= image_height)) {
        return;
    }
    const auto pixel_idx = j * image_width + i;
    color pixel_color(0, 0, 0);
    for (int s = 0; s < samples_per_pixel; ++s) {
        auto u = (i + random_double(&rand_states[pixel_idx])) / (image_width - 1);
        auto v = (j + random_double(&rand_states[pixel_idx])) / (image_height - 1);
        ray r = cam.get_ray(u, v, &rand_states[pixel_idx]);
        pixel_color += ray_color(r, *world, max_depth, &rand_states[pixel_idx]);
    }
    pixel_color /= samples_per_pixel;
    pixel_color[0] = std::sqrt(pixel_color[0]);
    pixel_color[1] = std::sqrt(pixel_color[1]);
    pixel_color[2] = std::sqrt(pixel_color[2]);
    buffer[pixel_idx] = pixel_color;
}

__global__ void init_curand_state(hiprandState* rand_states, int max_x, int max_y, unsigned int seed) {
    const auto x = threadIdx.x + blockDim.x * blockIdx.x;
    const auto y = threadIdx.y + blockDim.y * blockIdx.y;
    // We may be launching more threads than necessary.
    // Ignore those threads.
    if ((x >= max_x) || (y >= max_y)) {
        return;
    }

    const auto idx = y * max_x + x;
    // More efficient than using same seed with different sequence number, according to NVIDIA docs: https://docs.nvidia.com/cuda/hiprand/device-api-overview.html#performance-notes)
    hiprand_init(seed + idx, 0, 0, &rand_states[idx]);
}

int main() {


    // Image

    const unsigned int seed = 5222;
    const auto aspect_ratio = 16.0 / 9.0;
    const int image_width = IMAGE_WIDTH;
    const int image_height = static_cast<int>(image_width / aspect_ratio);
    const int samples_per_pixel = SAMPLES_PER_PIXEL;
    const int max_depth = MAX_DEPTH;

    // World

    std::srand(seed);

    // NOTE: To ensure consistency across all types of implementations, namely,
    // serial, OpenMP, and CUDA, we fix the random seed and use the same random
    // number generator to generate the scene.
    // Since we cannot find a CUDA implementation of a random number generator
    // that uses the same algorithm as the CPU version (`rand`), we cannot
    // initialize the random number generator on the GPU. Thus, random numbers
    // are pre-generated on the CPU and copied to the GPU.

    // By using the seed 5222, the 4070 random numbers are required to generate the scene.
    constexpr std::size_t num_rand_nums = MAP_SIZE * MAP_SIZE * 9;
    int* rand_nums = nullptr;
    checkCudaErrors(hipMallocManaged(&rand_nums, sizeof(int) * num_rand_nums));
    for (std::size_t i = 0; i < num_rand_nums; ++i) {
        rand_nums[i] = std::rand();
    }

    // We have a single thread initialize the world.
    hittable_list* world = nullptr;
    checkCudaErrors(hipMallocAsync(&world, sizeof(hittable_list), 0));
    random_scene<<<1, 1>>>(world, rand_nums, num_rand_nums);

    // Camera

    point3 lookfrom(13,2,3);
    point3 lookat(0,0,0);
    vec3 vup(0,1,0);
    auto dist_to_focus = 10.0;
    auto aperture = 0.1;

    camera cam(lookfrom, lookat, vup, 20, aspect_ratio, aperture, dist_to_focus);

    // Divide the workload

    auto block_size = dim3(CUDA_BLOCK_SIZE, CUDA_BLOCK_SIZE);
    // Round up the grid size to make sure we have enough threads.
    auto grid_size =
        dim3((image_width + block_size.x - 1) / block_size.x,
             (image_height + block_size.y - 1) / block_size.y);

    // Prepare random number generator to be used in the kernel function

    hiprandState* rand_states = nullptr;
    checkCudaErrors(hipMalloc(&rand_states, sizeof(hiprandState) * image_width * image_height));
    init_curand_state<<<grid_size, block_size>>>(rand_states, image_width, image_height, seed);

    // Render

    // The buffer is used by both CPU and GPU.
    vec3* buffer = nullptr;
    checkCudaErrors(hipMallocManaged(&buffer, sizeof(vec3) * image_width * image_height));
    render<<<grid_size, block_size>>>(buffer, image_width, image_height, world, cam, max_depth, samples_per_pixel, rand_states);

    // Conclude all device work before reading them out.
    checkCudaErrors(hipDeviceSynchronize());

    // Write color

    std::cout << "P3\n" << image_width << ' ' << image_height << "\n255\n";
    for (int j = image_height-1; j >= 0; --j) {
        std::cerr << "\rScanlines remaining: " << j << ' ' << std::flush;
        for (int i = 0; i < image_width; ++i) {
            const auto& pixel_color = buffer[j * image_width + i];
            std::cout << static_cast<int>(256 * clamp(pixel_color.x(), 0.0, 0.999)) << ' '
                      << static_cast<int>(256 * clamp(pixel_color.y(), 0.0, 0.999)) << ' '
                      << static_cast<int>(256 * clamp(pixel_color.z(), 0.0, 0.999)) << '\n';
        }
    }
    std::cerr << "\nDone.\n";

    checkCudaErrors(hipFree(buffer));
    checkCudaErrors(hipFree(rand_states));
    checkCudaErrors(hipFree(world));
    checkCudaErrors(hipFree(rand_nums));
}
